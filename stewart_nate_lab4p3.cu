/*
 * Name: Nate Steawrt
 * Date: 04-04-16
 * Description: Serial implementation of Matrix multiplication with transpose 
 */


#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define RANDOM_VALUE_MIN 1
#define RANDOM_VALUE_MAX 1000

#define MATRIX_DIM 16 

/*
 * Calculate and return a random value between min and max.
 */
int randInt(int min, int max) {
	return rand() % max + min;
}

/*
 * Output the matrix to fout
 */
void outputMatrix(FILE *fout, int *matrix, int rows, int cols) {
	int i, j;

	for (i = 0; i < rows; i++) {
		for (j = 0; j < cols; j++) {
			fprintf(fout,  "%d ", *(matrix + i * cols + j));
		}
		fprintf(fout, "\n");
	}
}

__global__ void computeMath(int *matrix) {

	// Grab the two indices dependent on the block/thread structure
	int col = blockIdx.x;
	int row = blockIdx.y * blockDim.x + threadIdx.x;

	// Only transpose if the column id is greater than the row id
	if (col > row) {
		int *transpose = matrix + col * MATRIX_DIM + row;
		int *result = matrix + row * MATRIX_DIM + col;
		int temp = *transpose;
		*transpose = *result;
		*result = temp;
	}

}

/*
 * Check if an error occurred during the last CUDA command
 */
void checkError() {
	int errorCode = hipGetLastError();

	if (errorCode != 0) {
		printf("Error %d occurred during last operation.\n", errorCode);
	}
}

int main(void) {

	// Declare the needed variables
	int i, j;
	
	// Variables for timing
	time_t startTime, endTime;
	clock_t clockTime;

	// Seed the random number generator
	srand(time(NULL));

	// Define thread hierarchy
	int nblocksX = 16;
	int nblocksY = 4;
	int dimX = 4;

	// Declare the memory pointers
	int *h_matrix, *d_matrix;

	// Allocate memory for host and device
	size_t memSize = MATRIX_DIM * MATRIX_DIM * sizeof(*h_matrix);

	// Create space on the host and device for matrix
	h_matrix = (int *)malloc(memSize);
	hipMalloc( (void**) &d_matrix, memSize);
	checkError();

	// Initialize the array
	int *m_ptr = h_matrix; // Setup a traversal pointer for the matrix
	for (i = 0; i < MATRIX_DIM; i++) {
		for (j = 0; j < MATRIX_DIM; j++, m_ptr++) {
			*m_ptr = j; randInt(RANDOM_VALUE_MIN, RANDOM_VALUE_MAX);
		}
	}
	hipMemcpy(d_matrix, h_matrix, memSize, hipMemcpyHostToDevice);
	checkError();

	// Set up grid and block structure
	dim3 dimGrid(nblocksX, nblocksY);
	dim3 dimBlock(dimX);

	outputMatrix(stdout, h_matrix, MATRIX_DIM, MATRIX_DIM);

	// Launch the kernel and begin timer	
	time(&startTime);
	clockTime = clock();
	computeMath<<< dimGrid, dimBlock >>>(d_matrix);
	
	// Stop timer and retrieve results
	hipMemcpy(h_matrix, d_matrix, memSize, hipMemcpyDeviceToHost);
	checkError();
	time(&endTime);
	clockTime = clock() - clockTime;

	outputMatrix(stdout, h_matrix, MATRIX_DIM, MATRIX_DIM);

	long numFloatingPointOperations = MATRIX_DIM * MATRIX_DIM;
	double gflops = numFloatingPointOperations / ((double)clockTime/1000000) / 1000000000;
	printf("*********************************************************************\n");
	printf("Number of floating point operations:%ld\n", numFloatingPointOperations);
	printf("Estimated GFlops:%lf GFlops\n\n", gflops);
	printf("elapsed convergence loop time\t(clock): %lu\n", clockTime);
	printf("elapsed convergence loop time\t (time): %.f\n", difftime(endTime, startTime));
	printf("*********************************************************************\n");

	free(h_matrix);
	hipFree(d_matrix);
	checkError();
}

