/*
 * Name: Nate Steawrt
 * Date: 04-04-16
 * Description: Serial implementation of Matrix morphism
 */


#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define RANDOM_VALUE_MIN 1.0
#define RANDOM_VALUE_MAX 2.0

#define NUM_ROWS 4097
#define NUM_COLS 4097

/*
 * Calculate and return a random value between min and max.
 */
double randDouble(double min, double max) {
	double range =  max - min;
	double dist = RAND_MAX / range;
	return min + (rand() / dist);
}

/*
 * Output the matrix to fout
 */
void outputMatrix(FILE *fout, double *matrix, int rows, int cols) {
	int i, j;

	for (i = 0; i < rows; i++) {
		for (j = 0; j < cols; j++) {
			fprintf(fout,  "%lf ", *(matrix + i * cols + j));
		}
		fprintf(fout, "\n");
	}
}

__global__ void computeMath(double *matrix) {
    
    int i;

    // Grab id of thread
    int threadId = blockDim.x * threadIdx.y + threadIdx.x + 1;
    
    // Declare pointers to the two arguments of the addition and the result pointer
	double *f_ptr, *first_ptr, *second_ptr;
	
	// Grab starting points for pointers
    f_ptr = matrix + threadId * NUM_COLS;
    first_ptr = matrix + (threadId - 1) * NUM_COLS + 1;
    second_ptr = f_ptr + 1;
    
    // Compute a single row
    for (i = 0; i < NUM_COLS - 1; i++, f_ptr++, first_ptr++, second_ptr++) {
        *f_ptr = *first_ptr + *second_ptr;
    }
}

/*
 * Check if an error occurred during the last CUDA command
 */
void checkError() {
	int errorCode = hipGetLastError();

	if (errorCode != 0) {
		printf("Error %d occurred during last operation.\n", errorCode);
	}
}

int main(void) {

	// Declare the needed variables
	int i, j;
	
	// Variables for timing
	time_t startTime, endTime;
	clock_t clockTime;

	// Seed the random number generator
	srand(time(NULL));
    
    // Define thread hierarchy
    int nblocks = 16;
    int dimX = 128;
    int dimY = 2;

    // Declare the memory pointers
    double *h_matrix, *d_matrix;
    
    // Allocate memory for host and device
    size_t memSize = NUM_ROWS * NUM_COLS * sizeof(*h_matrix);
	
	// Create space on the host and device for matrix
    h_matrix = (double *)malloc(memSize);
    hipMalloc( (void**) &d_matrix, memSize);
	checkError();

	// Initialize the matrix and copy values into device
	double *f_ptr = h_matrix; // Setup a traversal pointer
	for (i = 0; i < NUM_ROWS; i++) {
		for (j = 0; j < NUM_COLS; j++, f_ptr++) {
			*f_ptr = randDouble(RANDOM_VALUE_MIN, RANDOM_VALUE_MAX);
		}
	}
    hipMemcpy(d_matrix, h_matrix, memSize, hipMemcpyHostToDevice);
	checkError();
    
    // Set up grid and block structure
    dim3 dimGrid(nblocks);
    dim3 dimBlock(dimX, dimY);
    
    // Launch the kernel and begin timer 
    time(&startTime);
	clockTime = clock();
    for (i = 0; i < 100; i++) {
		computeMath<<< dimGrid, dimBlock >>>(d_matrix);
		checkError();
	}
    
    // stop timer and retrieve results
    hipMemcpy(h_matrix, d_matrix, memSize, hipMemcpyDeviceToHost);
	checkError();
    time(&endTime);
	clockTime = clock() - clockTime;
    
	unsigned long long numFloatingPointOperations = 100 * (NUM_ROWS-1) * (NUM_COLS-1);
	double gflops = numFloatingPointOperations / ((double)clockTime/1000000) / 1000000000;
	printf("*********************************************************************\n");
	printf("Number of floating point operations:%ld\n", numFloatingPointOperations);
	printf("Estimated GFlops:%lf GFlops\n\n", gflops);
	printf("elapsed convergence loop time\t(clock): %lu\n", clockTime);
	printf("elapsed convergence loop time\t (time): %.f\n", difftime(endTime, startTime));
	printf("*********************************************************************\n");

	free(h_matrix);
    hipFree(d_matrix);
	checkError();
}

